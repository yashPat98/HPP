#include "hip/hip_runtime.h"
// --- Headers ---
#include <hip/hip_runtime.h>
#include <stdio.h>

// --- Macros ---
#define CHUNK    (1024 * 1024)
#define SIZE     (CHUNK * 20)

// --- Variable Declaration ---
int *hostInputA = NULL;
int *hostInputB = NULL;
int *hostOutput = NULL;

int *deviceInputA0 = NULL;
int *deviceInputB0 = NULL;
int *deviceOutput0 = NULL;

int *deviceInputA1 = NULL;
int *deviceInputB1 = NULL;
int *deviceOutput1 = NULL;

hipEvent_t start, stop;
hipStream_t stream0, stream1;

// --- CUDA KERNEL DEFINITION ---
__global__ void kernel(int *inputA, int *inputB, int *output)
{
    //variable declaration
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    //code
    if(tid < CHUNK)
    {
        int tid1 = (tid + 1) % 256;
        int tid2 = (tid + 2) % 256;

        float inputA_avg = (inputA[tid] + inputA[tid1] + inputA[tid2]) / 3.0f;
        float inputB_avg = (inputB[tid] + inputB[tid1] + inputB[tid2]) / 3.0f;
        
        output[tid] = (inputA_avg + inputB_avg) / 2;
    }
}

// --- main() ---
int main(void)
{
    //function declaration
    void cleanup(void);

    //variable declaration
    hipError_t err = hipSuccess;
    hipDeviceProp_t prop;
    int deviceID;
    float elapsedTime;
    
    //code
    //check for device overlap capability
    err = hipGetDevice(&deviceID);
    if(err != hipSuccess)
    {
        printf("GPU Error - hipGetDevice() failed : %s.\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipGetDeviceProperties(&prop, deviceID);
    if(err != hipSuccess)
    {
        printf("GPU Error - hipGetDeviceProperties() failed : %s.\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    if(!prop.deviceOverlap)
    {
        printf("Device will not handle overlaps, so no speed up from streams\n");
        return (0);
    }

    //create cuda events
    err = hipEventCreate(&start);
    if(err != hipSuccess)
    {
        printf("GPU Error - hipEventCreate() failed for start: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipEventCreate(&stop);
    if(err != hipSuccess)
    {
        printf("GPU Error - hipEventCreate() failed for stop : %s\n", hipGetErrorString(err));
        hipEventDestroy(start);
        exit(EXIT_FAILURE);
    }

    //initialize the stream0 & stream1
    err = hipStreamCreate(&stream0);
    if(err != hipSuccess)
    {
        printf("GPU Error - hipStreamCreate() failed for stream0 : %s\n", hipGetErrorString(err));
        cleanup();
        exit(EXIT_FAILURE);
    }    

    err = hipStreamCreate(&stream1);
    if(err != hipSuccess)
    {
        printf("GPU Error - hipStreamCreate() failed for stream1 : %s\n", hipGetErrorString(err));
        cleanup();
        exit(EXIT_FAILURE);
    } 

    //allocate host memory (page-locked)
    err = hipHostAlloc((void **)&hostInputA, SIZE * sizeof(int), hipHostMallocDefault);
    if(err != hipSuccess)
    {
        printf("CPU Memory Fatal Error - hipHostAlloc() failed for hostInputA : %s.\n", hipGetErrorString(err));
        cleanup();
        exit(EXIT_FAILURE);
    }

    err = hipHostAlloc((void **)&hostInputB, SIZE * sizeof(int), hipHostMallocDefault);
    if(err != hipSuccess)
    {
        printf("CPU Memory Fatal Error - hipHostAlloc() failed for hostInputB : %s.\n", hipGetErrorString(err));
        cleanup();
        exit(EXIT_FAILURE);
    }

    err = hipHostAlloc((void **)&hostOutput, SIZE * sizeof(int), hipHostMallocDefault);
    if(err != hipSuccess)
    {
        printf("CPU Memory Fatal Error - hipHostAlloc() failed for hostOutput : %s.\n", hipGetErrorString(err));
        cleanup();
        exit(EXIT_FAILURE);
    }

    //allocate device memory for stream 0
    err = hipMalloc((void **)&deviceInputA0, CHUNK * sizeof(int));
    if(err != hipSuccess)
    {
        printf("GPU Memory Fatal Error - hipMalloc() failed for deviceInputA0 : %s.\n", hipGetErrorString(err));
        cleanup();
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **)&deviceInputB0, CHUNK * sizeof(int));
    if(err != hipSuccess)
    {
        printf("GPU Memory Fatal Error - hipMalloc() failed for deviceInputB0 : %s.\n", hipGetErrorString(err));
        cleanup();
        exit(EXIT_FAILURE);
    }    

    err = hipMalloc((void **)&deviceOutput0, CHUNK * sizeof(int));
    if(err != hipSuccess)
    {
        printf("GPU Memory Fatal Error - hipMalloc() failed for deviceOutput0 : %s.\n", hipGetErrorString(err));
        cleanup();
        exit(EXIT_FAILURE);
    }    

    //allocate device memory for stream 1
    err = hipMalloc((void **)&deviceInputA1, CHUNK * sizeof(int));
    if(err != hipSuccess)
    {
        printf("GPU Memory Fatal Error - hipMalloc() failed for deviceInputA1 : %s.\n", hipGetErrorString(err));
        cleanup();
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **)&deviceInputB1, CHUNK * sizeof(int));
    if(err != hipSuccess)
    {
        printf("GPU Memory Fatal Error - hipMalloc() failed for deviceInputB1 : %s.\n", hipGetErrorString(err));
        cleanup();
        exit(EXIT_FAILURE);
    }    

    err = hipMalloc((void **)&deviceOutput1, CHUNK * sizeof(int));
    if(err != hipSuccess)
    {
        printf("GPU Memory Fatal Error - hipMalloc() failed for deviceOutput1 : %s.\n", hipGetErrorString(err));
        cleanup();
        exit(EXIT_FAILURE);
    }

    //fill the host input memory
    for(int i = 0; i < SIZE; i++)
    {
        hostInputA[i] = (float)((1.0f / (float)RAND_MAX) * rand());
        hostInputB[i] = (float)((1.0f / (float)RAND_MAX) * rand());
    }

    //start timer 
    err = hipEventRecord(start, 0);
    if(err != hipSuccess)
    {
        printf("GPU Error - hipEventRecord() failed for start : %s.\n", hipGetErrorString(err));
        cleanup();
        exit(EXIT_FAILURE);
    }

    //cuda kernel configuration
    dim3 DimGrid = dim3(CHUNK / 256, 1, 1);
    dim3 DimBlock = dim3(256, 1, 1);

    //now loop over full data, in bite-sized chunks
    for(int i = 0; i < SIZE; i += (CHUNK * 2))
    {
        //stream 0
        //copy the locked memory to the device, async
        err = hipMemcpyAsync(deviceInputA0, hostInputA + i, CHUNK * sizeof(int), hipMemcpyHostToDevice, stream0);
        if(err != hipSuccess)
        {
            printf("GPU Error - hipMemcpyAsync() failed for Input1 : %s.\n", hipGetErrorString(err));
            cleanup();
            exit(EXIT_FAILURE);
        }

        err = hipMemcpyAsync(deviceInputB0, hostInputB + i, CHUNK * sizeof(int), hipMemcpyHostToDevice, stream0);
        if(err != hipSuccess)
        {
            printf("GPU Error - hipMemcpyAsync() failed for Input2 : %s.\n", hipGetErrorString(err));
            cleanup();
            exit(EXIT_FAILURE);
        }

        kernel<<<DimGrid, DimBlock, 0, stream0>>>(deviceInputA0, deviceInputB0, deviceOutput0);

        //copy the data from device to locked memory
        err = hipMemcpyAsync(hostOutput + i, deviceOutput0, CHUNK * sizeof(int), hipMemcpyDeviceToHost, stream0);
        if(err != hipSuccess)
        {
            printf("GPU Error - hipMemcpyAsync() failed for Output : %s.\n", hipGetErrorString(err));
            cleanup();
            exit(EXIT_FAILURE);
        }

        //stream 1
        //copy the locked memory to the device, async
        err = hipMemcpyAsync(deviceInputA1, hostInputA + i + CHUNK, CHUNK * sizeof(int), hipMemcpyHostToDevice, stream1);
        if(err != hipSuccess)
        {
            printf("GPU Error - hipMemcpyAsync() failed for Input1 : %s.\n", hipGetErrorString(err));
            cleanup();
            exit(EXIT_FAILURE);
        }

        err = hipMemcpyAsync(deviceInputB1, hostInputB + i + CHUNK, CHUNK * sizeof(int), hipMemcpyHostToDevice, stream1);
        if(err != hipSuccess)
        {
            printf("GPU Error - hipMemcpyAsync() failed for Input2 : %s.\n", hipGetErrorString(err));
            cleanup();
            exit(EXIT_FAILURE);
        }

        kernel<<<DimGrid, DimBlock, 0, stream1>>>(deviceInputA1, deviceInputB1, deviceOutput1);

        //copy the data from device to locked memory
        err = hipMemcpyAsync(hostOutput + i + CHUNK, deviceOutput1, CHUNK * sizeof(int), hipMemcpyDeviceToHost, stream1);
        if(err != hipSuccess)
        {
            printf("GPU Error - hipMemcpyAsync() failed for Output : %s.\n", hipGetErrorString(err));
            cleanup();
            exit(EXIT_FAILURE);
        }
    }

    err = hipStreamSynchronize(stream0);
    if(err != hipSuccess)
    {
        printf("GPU Error - hipStreamSynchronize() failed : %s.\n", hipGetErrorString(err));
        cleanup();
        exit(EXIT_FAILURE);
    }

    err = hipStreamSynchronize(stream1);
    if(err != hipSuccess)
    {
        printf("GPU Error - hipStreamSynchronize() failed : %s.\n", hipGetErrorString(err));
        cleanup();
        exit(EXIT_FAILURE);
    }

    //stop timer
    err = hipEventRecord(stop, 0);
    if(err != hipSuccess)
    {
        printf("GPU Error - hipEventRecord() failed for stop : %s.\n", hipGetErrorString(err));
        cleanup();
        exit(EXIT_FAILURE);
    }

    err = hipEventSynchronize(stop);
    if(err != hipSuccess)
    {
        printf("GPU Error - hipEventSynchronize() failed for stop : %s.\n", hipGetErrorString(err));
        cleanup();
        exit(EXIT_FAILURE);
    }

    err = hipEventElapsedTime(&elapsedTime, start, stop);
    if(err != hipSuccess)
    {
        printf("GPU Error - hipEventElapsedTime() failed : %s.\n", hipGetErrorString(err));
        cleanup();
        exit(EXIT_FAILURE);
    }    

    printf("Time taken for double stream : %3.1f ms\n", elapsedTime);

    //total cleanup
    cleanup();

    return (0);
}

void cleanup(void)
{
    //code
    //free device memory for stream 1
    if(deviceOutput1)
    {
        hipFree(deviceOutput1);
        deviceOutput1 = NULL;
    }

    if(deviceInputB1)
    {
        hipFree(deviceInputB1);
        deviceInputB1 = NULL;
    }

    if(deviceInputA1)
    {
        hipFree(deviceInputA1);
        deviceInputA1 = NULL;
    }

    //free device memory for stream 0
    if(deviceOutput0)
    {
        hipFree(deviceOutput0);
        deviceOutput0 = NULL;
    }

    if(deviceInputB0)
    {
        hipFree(deviceInputB0);
        deviceInputB0 = NULL;
    }

    if(deviceInputA0)
    {
        hipFree(deviceInputA0);
        deviceInputA0 = NULL;
    }

    //free host memory 
    if(hostOutput)
    {
        hipHostFree(hostOutput);
        hostOutput = NULL;
    }

    if(hostInputB)
    {
        hipHostFree(hostInputB);
        hostInputB = NULL;
    }

    if(hostInputA)
    {
        hipHostFree(hostInputA);
        hostInputA = NULL;
    }

    //destroy stream
    hipStreamDestroy(stream0);
    hipStreamDestroy(stream1);

    //destroy events
    hipEventDestroy(stop);
    hipEventDestroy(start);
}
