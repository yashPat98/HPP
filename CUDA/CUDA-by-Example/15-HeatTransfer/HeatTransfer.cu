#include "hip/hip_runtime.h"
// --- Headers ---
#include <hip/hip_runtime.h>
#include "../common/book.h"
#include "../common/cpu_anim.h"

// --- Macros ---
#define DIM       1024
#define PI        3.14159265f
#define MAX_TEMP  1.0f
#define MIN_TEMP  0.0001f
#define SPEED     0.25f

// --- Global Variables ---
struct DataBlock
{
    unsigned char   *output_bitmap;
    float           *device_inSrc;
    float           *device_outSrc;
    float           *device_constSrc;
    CPUAnimBitmap   *bitmap;
    
    hipEvent_t      start, stop;
    float            totalTime;
    float            frames;
};

float *hostGrid = NULL;

texture<float>  texConstSrc;
texture<float>  texIn;
texture<float>  texOut;

// --- CUDA KERNEL DEFINITION ---
__global__ void copy_const_kernel(float *input)
{
    //variable declaration
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int offset = x + y * blockDim.x * gridDim.x;

    float c = tex1Dfetch(texConstSrc, offset);

    //code
    if(c != 0)
        input[offset] = c;
}

__global__ void blend_kernel(float *dst, bool dstOut)
{
    //variable declaration
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int offset = x + y * blockDim.x * gridDim.x;

    int left = offset - 1;
    int right = offset + 1;
    int top = offset - DIM;
    int bottom = offset + DIM;

    float t, l, c, r, b;
    
    //code
    //check for out of bound index
    if(x == 0)
        left++;

    if(x == (DIM - 1))
        right--;

    if(y == 0)
        top += DIM;

    if(y == (DIM - 1))
        bottom -= DIM;

    //update the input temperature grid 
    if(dstOut)
    {
        t = tex1Dfetch(texIn, top);
        l = tex1Dfetch(texIn, left);
        c = tex1Dfetch(texIn, offset);
        r = tex1Dfetch(texIn, right);
        b = tex1Dfetch(texIn, bottom);
    }
    else
    {
        t = tex1Dfetch(texOut, top);
        l = tex1Dfetch(texOut, left);
        c = tex1Dfetch(texOut, offset);
        r = tex1Dfetch(texOut, right);
        b = tex1Dfetch(texOut, bottom);
    }

    dst[offset] = c + SPEED * (t + b + r + l - 4 * c);
}

// --- main() ---
int main(void)
{
    //function declaration
    void anim_gpu(DataBlock *d, int ticks);
    void cleanup(DataBlock *data);

    //variable declaration
    DataBlock data;
    CPUAnimBitmap bitmap(DIM, DIM, &data);
    hipError_t err = hipSuccess;

    //code
    data.bitmap     = &bitmap;
    data.totalTime  = 0.0f;
    data.frames     = 0.0f;

    //create cuda events 
    err = hipEventCreate(&data.start);
    if(err != hipSuccess)
    {
        printf("GPU Error - hipEventCreate() failed : %s.\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipEventCreate(&data.stop);
    if(err != hipSuccess)
    {
        printf("GPU Error - hipEventCreate() failed : %s.\n", hipGetErrorString(err));
        hipEventDestroy(data.start);
        exit(EXIT_FAILURE);
    }

    //allocate device memory
    err = hipMalloc((void **)&data.output_bitmap, bitmap.image_size());
    if(err != hipSuccess)
    {
        printf("GPU Memory Fatal Error - hipMalloc() failed : %s.\n", hipGetErrorString(err));
        cleanup(&data);
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **)&data.device_inSrc, bitmap.image_size());
    if(err != hipSuccess)
    {
        printf("GPU Memory Fatal Error - hipMalloc() failed : %s.\n", hipGetErrorString(err));
        cleanup(&data);
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **)&data.device_outSrc, bitmap.image_size());
    if(err != hipSuccess)
    {
        printf("GPU Memory Fatal Error - hipMalloc() failed : %s.\n", hipGetErrorString(err));
        cleanup(&data);
        exit(EXIT_FAILURE);
    }
    
    err = hipMalloc((void **)&data.device_constSrc, bitmap.image_size());
    if(err != hipSuccess)
    {
        printf("GPU Memory Fatal Error - hipMalloc() failed : %s.\n", hipGetErrorString(err));
        cleanup(&data);
        exit(EXIT_FAILURE);
    }
    
    //bind textures
    err = hipBindTexture(NULL, texConstSrc, data.device_constSrc, bitmap.image_size());
    if(err != hipSuccess)
    {
        printf("GPU Error - hipBindTexture() failed : %s.\n", hipGetErrorString(err));
        cleanup(&data);
        exit(EXIT_FAILURE);
    }

    err = hipBindTexture(NULL, texIn, data.device_inSrc, bitmap.image_size());
    if(err != hipSuccess)
    {
        printf("GPU Error - hipBindTexture() failed : %s.\n", hipGetErrorString(err));
        cleanup(&data);
        exit(EXIT_FAILURE);
    }

    err = hipBindTexture(NULL, texOut, data.device_outSrc, bitmap.image_size());
    if(err != hipSuccess)
    {
        printf("GPU Error - hipBindTexture() failed : %s.\n", hipGetErrorString(err));
        cleanup(&data);
        exit(EXIT_FAILURE);
    }

    //allocate host memory 
    hostGrid = (float *)malloc(bitmap.image_size());
    if(hostGrid == NULL)
    {
        printf("CPU Memory Fatal Error - can not allocate enough memory for grid.\n");
        cleanup(&data);
        exit(EXIT_FAILURE);
    }

    //initialize the constant data
    for(int i = 0; i < DIM * DIM; i++)
    {
        int x = i % DIM;
        int y = i / DIM;

        hostGrid[i] = 0;

        if((x > 300) && (x < 600) && (y > 310) && (y < 601))
            hostGrid[i] = MAX_TEMP;
    }

    hostGrid[DIM * 100 + 100] = (MAX_TEMP + MIN_TEMP) / 2.0f;
    hostGrid[DIM * 700 + 100] = MIN_TEMP;
    hostGrid[DIM * 300 + 300] = MIN_TEMP;
    hostGrid[DIM * 200 + 700] = MIN_TEMP;

    for(int y = 800; y < 900; y++)
    {
        for(int x = 400; x < 500; x++)
        {
            hostGrid[x + y * DIM] = MIN_TEMP;
        }
    }

    //copy the grid memory from host to device
    err = hipMemcpy(data.device_constSrc, hostGrid, bitmap.image_size(), hipMemcpyHostToDevice);
    if(err != hipSuccess)
    {
        printf("GPU Memory Fatal Error - hipMemcpy() failed : %s.\n", hipGetErrorString(err));
        cleanup(&data);
        exit(EXIT_FAILURE);
    }

    //initialize the input data
    for(int y = 800; y < DIM; y++)
    {
        for(int x = 0; x < 200; x++)
        {
            hostGrid[x + y * DIM] = MAX_TEMP;
        }
    }

    err = hipMemcpy(data.device_inSrc, hostGrid, bitmap.image_size(), hipMemcpyHostToDevice);
    if(err != hipSuccess)
    {
        printf("GPU Memory Fatal Error - hipMemcpy() failed : %s.\n", hipGetErrorString(err));
        cleanup(&data);
        exit(EXIT_FAILURE);
    }
    
    bitmap.anim_and_exit((void (*)(void *, int))anim_gpu, (void (*)(void *))cleanup);
}

void anim_gpu(DataBlock *data, int ticks)
{
    //function declaration
    void cleanup(DataBlock *data);

    //variable declaration
    CPUAnimBitmap *bitmap = data->bitmap;
    volatile bool dstOut = true;
    float elapsedTime = 0.0f;
    hipError_t err = hipSuccess;
    
    //code
    //start timer
    err = hipEventRecord(data->start, 0);
    if(err != hipSuccess)
    {
        printf("GPU Error - hipEventRecord() failed : %s.\n", hipGetErrorString(err));
        cleanup(data);
        exit(EXIT_FAILURE);
    }

    //kernel configuration
    dim3 DimGrid = dim3(DIM / 16, DIM / 16);
    dim3 DimBlock = dim3(16, 16);

    for(int i = 0; i < 90; i++)
    {
        float *in = NULL;
        float *out = NULL;

        if(dstOut)
        {
            in = data->device_inSrc;
            out = data->device_outSrc;
        }
        else
        {
            out = data->device_inSrc;
            in = data->device_outSrc;
        }

        copy_const_kernel<<<DimGrid, DimBlock>>>(in);
        
        blend_kernel<<<DimGrid, DimBlock>>>(out, dstOut);

        dstOut = !dstOut;
    }

    float_to_color<<<DimGrid, DimBlock>>>(data->output_bitmap, data->device_inSrc);

    //copy back to host memory
    err = hipMemcpy(bitmap->get_ptr(), data->output_bitmap, bitmap->image_size(), hipMemcpyDeviceToHost);
    if(err != hipSuccess)
    {
        printf("GPU Memory Fatal Error - hipMemcpy() failed : %s.\n", hipGetErrorString(err));
        cleanup(data);
        exit(EXIT_FAILURE);
    }

    //stop timer
    err = hipEventRecord(data->stop, 0);
    if(err != hipSuccess)
    {
        printf("GPU Error - hipEventRecord() failed : %s.\n", hipGetErrorString(err));
        cleanup(data);
        exit(EXIT_FAILURE);
    }

    //synchronize
    err = hipEventSynchronize(data->stop);
    if(err != hipSuccess)
    {
        printf("GPU Error - hipEventSynchronize() failed : %s.\n", hipGetErrorString(err));
        cleanup(data);
        exit(EXIT_FAILURE);
    }

    err = hipEventElapsedTime(&elapsedTime, data->start, data->stop);
    if(err != hipSuccess)
    {
        printf("GPU Error - hipEventElapsedTime() failed : %s.\n", hipGetErrorString(err));
        cleanup(data);
        exit(EXIT_FAILURE);
    }

    data->totalTime += elapsedTime;
    data->frames = data->frames + 1;

    printf("Average Time Per Frame : %3.1f ms\n", data->totalTime / data->frames);
}

void cleanup(DataBlock *data)
{
    //code
    //free device memory
    if(data->device_inSrc)
    {
        hipFree(data->device_inSrc);
        data->device_inSrc = NULL;
    }

    if(data->device_outSrc)
    {
        hipFree(data->device_outSrc);
        data->device_outSrc = NULL;
    }

    if(data->device_constSrc)
    {
        hipFree(data->device_constSrc);
        data->device_constSrc = NULL;
    }

    //free host memory 
    if(hostGrid)
    {
        free(hostGrid);
        hostGrid = NULL;
    }

    //unbind textures
    hipUnbindTexture(texIn);
    hipUnbindTexture(texOut);
    hipUnbindTexture(texConstSrc);

    //deallocate events
    hipEventDestroy(data->start);
    hipEventDestroy(data->stop);
}
