//headers
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define N (33 * 1024)

//global variables declaration
int hostA[N];
int hostB[N];
int hostC[N];

int *deviceA;
int *deviceB;
int *deviceC;

// *** CUDA KERNEL DEFINITION ***
__global__ void add(int *a, int *b, int *c)
{
    //variable declaration
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    //code
    while(tid < N)
    {
        c[tid] = a[tid] + b[tid];
        tid += blockDim.x * gridDim.x;
    }
}

int main(int argc, char *argv[])
{
    //function declaration
    void cleanup(void);

    //code
    hipError_t err = hipSuccess;
    
    //allocate memory on device
    err = hipMalloc((void **)&deviceA, N * sizeof(int));
    if(err != hipSuccess)
    {
        printf("GPU Memory Fatal Error = %s In File Name %s At Line No %d.\nExiting Now ...\n", hipGetErrorString(err), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **)&deviceB, N * sizeof(int));
    if(err != hipSuccess)
    {
        printf("GPU Memory Fatal Error = %s In File Name %s At Line No %d.\nExiting Now ...\n", hipGetErrorString(err), __FILE__, __LINE__);
        cleanup();
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **)&deviceC, N * sizeof(int));
    if(err != hipSuccess)
    {
        printf("GPU Memory Fatal Error = %s In File Name %s At Line No %d.\nExiting Now ...\n", hipGetErrorString(err), __FILE__, __LINE__);
        cleanup();
        exit(EXIT_FAILURE);
    }

    //fill the host input array
    for(int i = 0; i < N; i++)
    {
        hostA[i] = i;
        hostB[i] = i * i;
    }

    //copy the host input array to device memory
    err = hipMemcpy(deviceA, hostA, (N * sizeof(int)), hipMemcpyHostToDevice);
    if(err != hipSuccess)
    {
        printf("GPU Memory Fatal Error = %s In File Name %s At Line No %d.\nExiting Now ...\n", hipGetErrorString(err), __FILE__, __LINE__);
        cleanup();
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(deviceB, hostB, (N * sizeof(int)), hipMemcpyHostToDevice);
    if(err != hipSuccess)
    {
        printf("GPU Memory Fatal Error = %s In File Name %s At Line No %d.\nExiting Now ...\n", hipGetErrorString(err), __FILE__, __LINE__);
        cleanup();
        exit(EXIT_FAILURE);
    }
    
    //cuda kernel configuration
    dim3 DimGrid = 128;
    dim3 DimBlock = 128;

    add<<<DimGrid, DimBlock>>>(deviceA, deviceB, deviceC);

    //copy output array back to host 
    err = hipMemcpy(hostC, deviceC, (N * sizeof(int)), hipMemcpyDeviceToHost);
    if(err != hipSuccess)
    {
        printf("GPU Memory Fatal Error = %s In File Name %s At Line No %d.\nExiting Now ...\n", hipGetErrorString(err), __FILE__, __LINE__);
        cleanup();
        exit(EXIT_FAILURE);
    }    

    //verify the results 
    bool success = true;
    for(int i = 0; i < N; i++)
    {
        if(hostA[i] + hostB[i] != hostC[i])
        {
            printf("Error : %d + %d != %d\n", hostA[i], hostB[i], hostC[i]);
            success = false;
        }
    }

    if(success)
    {
        printf("Addition Is Successful On GPU !\n");
    }

    //total cleanup
    cleanup();

    return (0);
}

void cleanup(void)
{
    if(deviceA)
    {
        hipFree(deviceA);
        deviceA = NULL;
    }

    if(deviceB)
    {
        hipFree(deviceB);
        deviceB = NULL;
    }

    if(deviceC)
    {
        hipFree(deviceC);
        deviceC = NULL;
    }
}
