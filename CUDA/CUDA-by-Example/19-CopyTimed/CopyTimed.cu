// --- Headers ---
#include <hip/hip_runtime.h>
#include "../common/book.h"

#define SIZE (64 * 1024 * 1024)
#define FAILURE -1

//--- Variable Declaration ---
int *host = NULL;
int *device = NULL;

hipEvent_t start, stop;
float elapsedTime;

// --- main() ---
int main(void)
{
    //function declaration
    float malloc_time(int size, bool up);
    float cudaHostAlloc_time(int size, bool up);

    //variable declaration
    float elapsedTime;
    float MB = (float)(100.0f * SIZE * sizeof(int) / (1024.0f * 1024.0f));

    //code
    //copy with malloc
    elapsedTime = malloc_time(SIZE, true);
    if(elapsedTime == FAILURE)
        exit(EXIT_FAILURE);

    printf("Time using malloc() : %3.1f ms\n", elapsedTime);
    printf("MB/s during copy up : %3.1f\n", MB/(elapsedTime / 1000));

    elapsedTime = malloc_time(SIZE, false);
    if(elapsedTime == FAILURE)
        exit(EXIT_FAILURE);

    printf("Time using malloc() : %3.1f ms\n", elapsedTime);
    printf("MB/s during copy down : %3.1f\n", MB/(elapsedTime / 1000));

    printf("\n\n");
    //copy with hipHostAlloc
    elapsedTime = cudaHostAlloc_time(SIZE, true);
    if(elapsedTime == FAILURE)
        exit(EXIT_FAILURE);

    printf("Time using hipHostAlloc() : %3.1f ms\n", elapsedTime);
    printf("MB/s during copy up : %3.1f\n", MB/(elapsedTime / 1000));

    elapsedTime = cudaHostAlloc_time(SIZE, false);
    if(elapsedTime == FAILURE)
        exit(EXIT_FAILURE);

    printf("Time using hipHostAlloc() : %3.1f ms\n", elapsedTime);
    printf("MB/s during copy down : %3.1f\n", MB/(elapsedTime / 1000));

    return (0);
}

float malloc_time(int size, bool up)
{
    //function declaration
    void cleanup_malloc(void);

    //variable declaration
    hipError_t err = hipSuccess;
    float elapsedTime;

    //code
    //create cuda events
    err = hipEventCreate(&start);
    if(err != hipSuccess)
    {
        printf("GPU Error - hipEventCreate() failed for start : %s.\n", hipGetErrorString(err));
        return (FAILURE);
    }

    err = hipEventCreate(&stop);
    if(err != hipSuccess)
    {
        printf("GPU Error - hipEventCreate() failed for stop : %s.\n", hipGetErrorString(err));
        hipEventDestroy(start);
        return (FAILURE);
    }

    //allocate host memory using malloc
    host = (int *)malloc(size * sizeof(int));
    if(host == NULL)
    {
        printf("CPU Memory Fatal Error - malloc() failed for host.\n");
        cleanup_malloc();
        return (FAILURE);
    }

    //allocate device memory 
    err = hipMalloc((void **)&device, sizeof(int) * size);
    if(err != hipSuccess)
    {
        printf("GPU Memory Fatal Error - hipMalloc() failed for device : %s.\n", hipGetErrorString(err));
        cleanup_malloc();
        return (FAILURE);
    }

    //start timer
    err = hipEventRecord(start, 0);
    if(err != hipSuccess)
    {
        printf("GPU Error - hipEventRecord() failed for start : %s.\n", hipGetErrorString(err));
        cleanup_malloc();
        return (FAILURE);
    }

    //copy
    for(int i = 0; i < 100; i++)
    {
        if(up)
        {
            err = hipMemcpy(device, host, size * sizeof(int), hipMemcpyHostToDevice);
            if(err != hipSuccess)
            {
                printf("GPU Error - hipMemcpy() failed for Host To Device : %s.\n", hipGetErrorString(err));
                cleanup_malloc();
                return (FAILURE);
            }
        }
        else
        {
            err = hipMemcpy(host, device, size * sizeof(int), hipMemcpyDeviceToHost);
            if(err != hipSuccess)
            {
                printf("GPU Error - hipMemcpy() failed for Device To Host : %s.\n", hipGetErrorString(err));
                cleanup_malloc();
                return (FAILURE);
            }
        }
    }

    //stop timer
    err = hipEventRecord(stop, 0);
    if(err != hipSuccess)
    {
        printf("GPU Error - hipEventRecord() failed for stop : %s.\n", hipGetErrorString(err));
        cleanup_malloc();
        return (FAILURE);
    }

    err = hipEventSynchronize(stop);
    if(err != hipSuccess)
    {
        printf("GPU Error - hipEventSynchronize() failed : %s.\n", hipGetErrorString(err));
        cleanup_malloc();
        return (FAILURE);
    }

    err = hipEventElapsedTime(&elapsedTime, start, stop);
    if(err != hipSuccess)
    {
        printf("GPU Error - hipEventElapsedTime() failed : %s.\n", hipGetErrorString(err));
        cleanup_malloc();
        return (FAILURE);
    }

    //total cleanup
    cleanup_malloc();

    return (elapsedTime);
}

float cudaHostAlloc_time(int size, bool up)
{
    //function declaration
    void cleanup_cudaHostAlloc(void);

    //variable declaration
    hipError_t err = hipSuccess;
    float elapsedTime;

    //code
    //create cuda events
    err = hipEventCreate(&start);
    if(err != hipSuccess)
    {
        printf("GPU Error - hipEventCreate() failed for start : %s.\n", hipGetErrorString(err));
        return (FAILURE);
    }

    err = hipEventCreate(&stop);
    if(err != hipSuccess)
    {
        printf("GPU Error - hipEventCreate() failed for stop : %s.\n", hipGetErrorString(err));
        hipEventDestroy(start);
        return (FAILURE);
    }

    //allocate host memory using hipHostAlloc
    err = hipHostAlloc((void **)&host, size * sizeof(int), hipHostMallocDefault);
    if(err != hipSuccess)
    {
        printf("GPU Memory Fatal Error - hipHostAlloc() failed for host : %s.\n", hipGetErrorString(err));
        cleanup_cudaHostAlloc();
        return (FAILURE);
    }

    //allocate device memory 
    err = hipMalloc((void **)&device, sizeof(int) * size);
    if(err != hipSuccess)
    {
        printf("GPU Memory Fatal Error - hipMalloc() failed for device : %s.\n", hipGetErrorString(err));
        cleanup_cudaHostAlloc();
        return (FAILURE);
    }

    //start timer
    err = hipEventRecord(start, 0);
    if(err != hipSuccess)
    {
        printf("GPU Error - hipEventRecord() failed for start : %s.\n", hipGetErrorString(err));
        cleanup_cudaHostAlloc();
        return (FAILURE);
    }

    //copy
    for(int i = 0; i < 100; i++)
    {
        if(up)
        {
            err = hipMemcpy(device, host, size * sizeof(int), hipMemcpyHostToDevice);
            if(err != hipSuccess)
            {
                printf("GPU Error - hipMemcpy() failed for Host To Device : %s.\n", hipGetErrorString(err));
                cleanup_cudaHostAlloc();
                return (FAILURE);
            }
        }
        else
        {
            err = hipMemcpy(host, device, size * sizeof(int), hipMemcpyDeviceToHost);
            if(err != hipSuccess)
            {
                printf("GPU Error - hipMemcpy() failed for Device To Host : %s.\n", hipGetErrorString(err));
                cleanup_cudaHostAlloc();
                return (FAILURE);
            }
        }
    }

    //stop timer
    err = hipEventRecord(stop, 0);
    if(err != hipSuccess)
    {
        printf("GPU Error - hipEventRecord() failed for stop : %s.\n", hipGetErrorString(err));
        cleanup_cudaHostAlloc();
        return (FAILURE);
    }

    err = hipEventSynchronize(stop);
    if(err != hipSuccess)
    {
        printf("GPU Error - hipEventSynchronize() failed : %s.\n", hipGetErrorString(err));
        cleanup_cudaHostAlloc();
        return (FAILURE);
    }

    err = hipEventElapsedTime(&elapsedTime, start, stop);
    if(err != hipSuccess)
    {
        printf("GPU Error - hipEventElapsedTime() failed : %s.\n", hipGetErrorString(err));
        cleanup_cudaHostAlloc();
        return (FAILURE);
    }

    //total cleanup
    cleanup_cudaHostAlloc();

    return (elapsedTime);
}

void cleanup_malloc(void)
{
    //code
    if(device)
    {
        hipFree(device);
        device = NULL;
    }

    if(host)
    {
        free(host);
        host = NULL;
    }

    hipEventDestroy(stop);
    hipEventDestroy(start);
}

void cleanup_cudaHostAlloc(void)
{
    //code
    if(device)
    {
        hipFree(device);
        device = NULL;
    }

    if(host)
    {
        hipHostFree(host);
        host = NULL;
    }

    hipEventDestroy(stop);
    hipEventDestroy(start);
}
