#include "hip/hip_runtime.h"
#include <iostream>
#include "../common/book.h"

__global__ void add(int a, int b, int *c)
{
    //code
    *c = a + b;
}

int main(void)
{
    //variable declaration
    int c;
    int *dev_c = NULL;

    //code 
    HANDLE_ERROR(hipMalloc((void**)&dev_c, sizeof(int)));

    add<<<1,1>>>(2, 7, dev_c);

    HANDLE_ERROR(hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost));

    printf("2 + 7 = %d\n", c);

    hipFree(dev_c);
    dev_c = NULL;

    return (0);
}
