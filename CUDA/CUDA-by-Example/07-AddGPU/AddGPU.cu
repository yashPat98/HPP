#include "hip/hip_runtime.h"
//headers
#include "../common/book.h"

#define N 10

int main(void)
{
    //function declaration
    __global__ void add(int *a, int *b, int *c);

    //variable declaration
    int i, a[N], b[N], c[N];
    int *dev_a = NULL;
    int *dev_b = NULL;
    int *dev_c = NULL;

    //allocate the memory on the GPU
    HANDLE_ERROR(hipMalloc((void**)&dev_a, N * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_b, N * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_c, N * sizeof(int)));

    //fill the arrays 'a' and 'b' on the CPU
    for(i = 0; i < N; i++)
    {
        a[i] = -i;
        b[i] = i * i;
    }

    //copy the arrays 'a' and 'b' to the GPU
    HANDLE_ERROR(hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice));

    add<<<N, 1>>>(dev_a, dev_b, dev_c);

    //copy the array 'c' back from the GPU to the CPU
    HANDLE_ERROR(hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost));

    //display the results
    for(i = 0; i < N; i++)
    {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    //free the memory
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    dev_a = NULL;
    dev_b = NULL;
    dev_c = NULL;

    return (0);
}

__global__ void add(int *a, int *b, int *c)
{
    //variable declaration
    int tid = blockIdx.x;               //handle the data at this index

    //code
    if(tid < N)
    {
        c[tid] = a[tid] + b[tid];
    }
}

