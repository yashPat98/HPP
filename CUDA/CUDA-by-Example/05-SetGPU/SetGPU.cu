#include "../common/book.h"

int main(void)
{
    //variable declaration
    hipDeviceProp_t prop;
    int dev;

    //code
    HANDLE_ERROR(hipGetDevice(&dev));
    printf("ID of current CUDA device : %d\n", dev);

    //zero out structure memory
    memset(&prop, 0, sizeof(hipDeviceProp_t));
    prop.major = 1;
    prop.minor = 3;

    HANDLE_ERROR(hipChooseDevice(&dev, &prop));
    printf("ID of CUDA device closest to revision 1.3 : %d\n", dev);

    HANDLE_ERROR(hipSetDevice(dev));

    return (0);
}