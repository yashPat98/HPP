#include "hip/hip_runtime.h"
// --- Headers ---
#include <hip/hip_runtime.h>
#include <stdio.h>

// --- Macros ---
#define CHUNK    (1024 * 1024)
#define SIZE     (CHUNK * 20)

// --- Variable Declaration ---
int *hostInput1 = NULL;
int *hostInput2 = NULL;
int *hostOutput = NULL;

int *deviceInput1 = NULL;
int *deviceInput2 = NULL;
int *deviceOutput = NULL;

hipEvent_t start, stop;
hipStream_t stream;

// --- CUDA KERNEL DEFINITION ---
__global__ void kernel(int *input1, int *input2, int *output)
{
    //variable declaration
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    //code
    if(tid < CHUNK)
    {
        int tid1 = (tid + 1) % 256;
        int tid2 = (tid + 2) % 256;

        float input1_avg = (input1[tid] + input1[tid1] + input1[tid2]) / 3.0f;
        float input2_avg = (input2[tid] + input2[tid1] + input2[tid2]) / 3.0f;
        
        output[tid] = (input1_avg + input2_avg) / 2;
    }
}

// --- main() ---
int main(void)
{
    //function declaration
    void cleanup(void);

    //variable declaration
    hipError_t err = hipSuccess;
    hipDeviceProp_t prop;
    int deviceID;
    float elapsedTime;
    
    //code
    //check for device overlap capability
    err = hipGetDevice(&deviceID);
    if(err != hipSuccess)
    {
        printf("GPU Error - hipGetDevice() failed : %s.\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipGetDeviceProperties(&prop, deviceID);
    if(err != hipSuccess)
    {
        printf("GPU Error - hipGetDeviceProperties() failed : %s.\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    if(!prop.deviceOverlap)
    {
        printf("Device will not handle overlaps, so no speed up from streams\n");
        return (0);
    }

    //create cuda events
    err = hipEventCreate(&start);
    if(err != hipSuccess)
    {
        printf("GPU Error - hipEventCreate() failed for start: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipEventCreate(&stop);
    if(err != hipSuccess)
    {
        printf("GPU Error - hipEventCreate() failed for stop : %s\n", hipGetErrorString(err));
        hipEventDestroy(start);
        exit(EXIT_FAILURE);
    }

    //initialize the stream
    err = hipStreamCreate(&stream);
    if(err != hipSuccess)
    {
        printf("GPU Error - hipStreamCreate() failed for stream : %s\n", hipGetErrorString(err));
        cleanup();
        exit(EXIT_FAILURE);
    }    

    //allocate host memory (page-locked)
    err = hipHostAlloc((void **)&hostInput1, SIZE * sizeof(int), hipHostMallocDefault);
    if(err != hipSuccess)
    {
        printf("CPU Memory Fatal Error - hipHostAlloc() failed for hostInput1 : %s.\n", hipGetErrorString(err));
        cleanup();
        exit(EXIT_FAILURE);
    }

    err = hipHostAlloc((void **)&hostInput2, SIZE * sizeof(int), hipHostMallocDefault);
    if(err != hipSuccess)
    {
        printf("CPU Memory Fatal Error - hipHostAlloc() failed for hostInput2 : %s.\n", hipGetErrorString(err));
        cleanup();
        exit(EXIT_FAILURE);
    }

    err = hipHostAlloc((void **)&hostOutput, SIZE * sizeof(int), hipHostMallocDefault);
    if(err != hipSuccess)
    {
        printf("CPU Memory Fatal Error - hipHostAlloc() failed for hostOutput : %s.\n", hipGetErrorString(err));
        cleanup();
        exit(EXIT_FAILURE);
    }

    //allocate device memory 
    err = hipMalloc((void **)&deviceInput1, CHUNK * sizeof(int));
    if(err != hipSuccess)
    {
        printf("GPU Memory Fatal Error - hipMalloc() failed for deviceInput1 : %s.\n", hipGetErrorString(err));
        cleanup();
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **)&deviceInput2, CHUNK * sizeof(int));
    if(err != hipSuccess)
    {
        printf("GPU Memory Fatal Error - hipMalloc() failed for deviceInput2 : %s.\n", hipGetErrorString(err));
        cleanup();
        exit(EXIT_FAILURE);
    }    

    err = hipMalloc((void **)&deviceOutput, CHUNK * sizeof(int));
    if(err != hipSuccess)
    {
        printf("GPU Memory Fatal Error - hipMalloc() failed for deviceOutput : %s.\n", hipGetErrorString(err));
        cleanup();
        exit(EXIT_FAILURE);
    }    

    //fill the host input memory
    for(int i = 0; i < SIZE; i++)
    {
        hostInput1[i] = (float)((1.0f / (float)RAND_MAX) * rand());
        hostInput2[i] = (float)((1.0f / (float)RAND_MAX) * rand());
    }

    //start timer 
    err = hipEventRecord(start, 0);
    if(err != hipSuccess)
    {
        printf("GPU Error - hipEventRecord() failed for start : %s.\n", hipGetErrorString(err));
        cleanup();
        exit(EXIT_FAILURE);
    }

    //cuda kernel configuration
    dim3 DimGrid = dim3(CHUNK / 256, 1, 1);
    dim3 DimBlock = dim3(256, 1, 1);

    //now loop over full data, in bite-sized chunks
    for(int i = 0; i < SIZE; i += CHUNK)
    {
        //copy the locked memory to the device, async
        err = hipMemcpyAsync(deviceInput1, hostInput1 + i, CHUNK * sizeof(int), hipMemcpyHostToDevice, stream);
        if(err != hipSuccess)
        {
            printf("GPU Error - hipMemcpyAsync() failed for Input1 : %s.\n", hipGetErrorString(err));
            cleanup();
            exit(EXIT_FAILURE);
        }

        err = hipMemcpyAsync(deviceInput2, hostInput2 + i, CHUNK * sizeof(int), hipMemcpyHostToDevice, stream);
        if(err != hipSuccess)
        {
            printf("GPU Error - hipMemcpyAsync() failed for Input2 : %s.\n", hipGetErrorString(err));
            cleanup();
            exit(EXIT_FAILURE);
        }

        kernel<<<DimGrid, DimBlock, 0, stream>>>(deviceInput1, deviceInput2, deviceOutput);

        //copy the data from device to locked memory
        err = hipMemcpyAsync(hostOutput + i, deviceOutput, CHUNK * sizeof(int), hipMemcpyDeviceToHost, stream);
        if(err != hipSuccess)
        {
            printf("GPU Error - hipMemcpyAsync() failed for Output : %s.\n", hipGetErrorString(err));
            cleanup();
            exit(EXIT_FAILURE);
        }
    }

    err = hipStreamSynchronize(stream);
    if(err != hipSuccess)
    {
        printf("GPU Error - hipStreamSynchronize() failed : %s.\n", hipGetErrorString(err));
        cleanup();
        exit(EXIT_FAILURE);
    }

    //stop timer
    err = hipEventRecord(stop, 0);
    if(err != hipSuccess)
    {
        printf("GPU Error - hipEventRecord() failed for stop : %s.\n", hipGetErrorString(err));
        cleanup();
        exit(EXIT_FAILURE);
    }

    err = hipEventSynchronize(stop);
    if(err != hipSuccess)
    {
        printf("GPU Error - hipEventSynchronize() failed for stop : %s.\n", hipGetErrorString(err));
        cleanup();
        exit(EXIT_FAILURE);
    }

    err = hipEventElapsedTime(&elapsedTime, start, stop);
    if(err != hipSuccess)
    {
        printf("GPU Error - hipEventElapsedTime() failed : %s.\n", hipGetErrorString(err));
        cleanup();
        exit(EXIT_FAILURE);
    }    

    printf("Time taken for single stream : %3.1f ms\n", elapsedTime);

    //total cleanup
    cleanup();

    return (0);
}

void cleanup(void)
{
    //code
    //free device memory 
    if(deviceOutput)
    {
        hipFree(deviceOutput);
        deviceOutput = NULL;
    }

    if(deviceInput2)
    {
        hipFree(deviceInput2);
        deviceInput2 = NULL;
    }

    if(deviceInput1)
    {
        hipFree(deviceInput1);
        deviceInput1 = NULL;
    }

    //free host memory 
    if(hostOutput)
    {
        hipHostFree(hostOutput);
        hostOutput = NULL;
    }

    if(hostInput2)
    {
        hipHostFree(hostInput2);
        hostInput2 = NULL;
    }

    if(hostInput1)
    {
        hipHostFree(hostInput1);
        hostInput1 = NULL;
    }

    //destroy stream
    hipStreamDestroy(stream);

    //destroy events
    hipEventDestroy(stop);
    hipEventDestroy(start);
}
