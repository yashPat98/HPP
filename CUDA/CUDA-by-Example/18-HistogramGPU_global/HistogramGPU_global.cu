#include "hip/hip_runtime.h"
// --- Headers ---
#include "../common/book.h"

#define SIZE (100 * 1024 * 1024)

// --- Variable Declaration ---
unsigned char *hostData        = NULL;
unsigned int *hostHistogram    = NULL;

unsigned char *deviceData      = NULL;
unsigned int *deviceHistogram  = NULL;

hipEvent_t start, stop;

// --- CUDA KERNEL DEFINITION ---
__global__ void HistogramKernel(unsigned char *data, long size, unsigned int *histogram)
{
    //variable declaration
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    //code 
    while(i < size)
    {
        atomicAdd(&histogram[data[i]], 1);
        i += stride;
    }
}

// --- main() ---
int main(void)
{
    //function declaration
    void cleanup(void);

    //variable declaration
    long histogramCount = 0;
    float elapsedTime;

    hipDeviceProp_t prop;
    hipError_t err = hipSuccess;
    
    //code
    //create cuda events
    err = hipEventCreate(&start);
    if(err != hipSuccess)
    {
        printf("GPU Error - hipEventCreate() failed for start : %s.\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    } 

    err = hipEventCreate(&stop);
    if(err != hipSuccess)
    {
        printf("GPU Error - hipEventCreate() failed for stop : %s.\n", hipGetErrorString(err));
        hipEventDestroy(start);
        exit(EXIT_FAILURE);
    }

    //allocate host memory
    hostData = (unsigned char *)big_random_block(SIZE);
    if(hostData == NULL)
    {
        printf("CPU Memory Fatal Error - malloc() failed for hostData.\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    hostHistogram = (unsigned int *)malloc(256 * sizeof(float));
    if(hostHistogram == NULL)
    {
        printf("CPU Memory Fatal Error - malloc() failed for hostHistogram.\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    //allocate device memory 
    err = hipMalloc((void **)&deviceData, SIZE);
    if(err != hipSuccess)
    {
        printf("GPU Memory Fatal Error - hipMalloc() failed for deviceData : %s.\n", hipGetErrorString(err));
        cleanup();
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **)&deviceHistogram, 256 * sizeof(float));
    if(err != hipSuccess)
    {
        printf("GPU Memory Fatal Error - hipMalloc() failed for deviceHistogram : %s.\n", hipGetErrorString(err));
        cleanup();
        exit(EXIT_FAILURE);
    }

    //zero-out host histogram memory
    for(int i = 0; i < 256; i++)
        hostHistogram[i] = 0;

    //zero-out device histogram memory
    err = hipMemset(deviceHistogram, 0, 256 * sizeof(float));
    if(err != hipSuccess)
    {
        printf("GPU Error - hipMemset() failed for deviceHistogram : %s.\n", hipGetErrorString(err));
        cleanup();
        exit(EXIT_FAILURE);
    }

    //copy data from host to device
    err = hipMemcpy(deviceData, hostData, SIZE, hipMemcpyHostToDevice);
    if(err != hipSuccess)
    {
        printf("GPU Error - hipMemcpy() failed for host to device : %s.\n", hipGetErrorString(err));
        cleanup();
        exit(EXIT_FAILURE);
    }



    // --- KERNEL CONFIGURATION ---
    //kernel launch - 2x the number of multi processors gave best timing
    err = hipGetDeviceProperties(&prop, 0);
    if(err != hipSuccess)
    {
        printf("GPU Error - cudaDeviceProperties() : %s.\n", hipGetErrorString(err));
        cleanup();
        exit(EXIT_FAILURE);
    }  

    int blocks = prop.multiProcessorCount;

    //start the timer
    err = hipEventRecord(start, 0);
    if(err != hipSuccess)
    {
        printf("GPU Error - hipEventRecord() failed for start : %s.\n", hipGetErrorString(err));
        cleanup();
        exit(EXIT_FAILURE);
    }

    //kernel launch
    HistogramKernel<<<blocks * 2, 256>>>(deviceData, SIZE, deviceHistogram);

    //copy histogram from device to host
    err = hipMemcpy(hostHistogram, deviceHistogram, 256 * sizeof(float), hipMemcpyDeviceToHost);
    if(err != hipSuccess)
    {
        printf("GPU Error - hipMemcpy() failed for device to host : %s.\n", hipGetErrorString(err));
        cleanup();
        exit(EXIT_FAILURE);
    }

    //stop the timer
    err = hipEventRecord(stop, 0);
    if(err != hipSuccess)
    {
        printf("GPU Error - hipEventRecord() failed for stop : %s.\n", hipGetErrorString(err));
        cleanup();
        exit(EXIT_FAILURE);
    }    

    //sync to stop event
    err = hipEventSynchronize(stop);
    if(err != hipSuccess)
    {
        printf("GPU Error - hipEventSynchronize() failed for stop : %s.\n", hipGetErrorString(err));
        cleanup();
        exit(EXIT_FAILURE);
    }

    //calculate time
    err = hipEventElapsedTime(&elapsedTime, start, stop);
    if(err != hipSuccess)
    {
        printf("GPU Error - hipEventElapsedTime() failed : %s.\n", hipGetErrorString(err));
        cleanup();
        exit(EXIT_FAILURE);
    }

    printf("Time to generate : %3.1f ms\n", elapsedTime);

    for(int i = 0; i < 256; i++)
        histogramCount += hostHistogram[i];

    printf("Histogram Sum : %ld\n", histogramCount);

    //verify that we have same count on CPU
    for(int i = 0; i < SIZE; i++)
        hostHistogram[hostData[i]]--;
    
    for(int i = 0; i < 256; i++)
        if(hostHistogram[i] != 0)
            printf("Failure at %d ! Off by %d\n", i, hostHistogram[i]);

    //total cleanup
    cleanup();

    return (0);
}

void cleanup(void)
{
    //code
    if(deviceHistogram)
    {
        hipFree(deviceHistogram);
        deviceHistogram = NULL;
    }

    if(deviceData)
    {
        hipFree(deviceData);
        deviceData = NULL;
    }

    if(hostHistogram)
    {
        free(hostHistogram);
        hostHistogram = NULL;
    }

    if(hostData)
    {
        free(hostData);
        hostData = NULL;
    }

    hipEventDestroy(stop);
    hipEventDestroy(start);
}
