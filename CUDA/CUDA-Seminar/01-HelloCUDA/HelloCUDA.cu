//headers
#include <stdio.h>     
#include <hip/hip_runtime.h>       //standard cuda header file

//global variables
int inputLength = 5;

float *hostInput1 = NULL;
float *hostInput2 = NULL;
float *hostOutput = NULL;

float *deviceInput1 = NULL;
float *deviceInput2 = NULL;
float *deviceOutput = NULL;

//global kernel function definition
__global__ void vecAdd(float *in1, float *in2, float *out, int len)
{
    //variable declaration
    //row * width + column
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    //code 
    if(i < len)
    {
        out[i] = in1[i] + in2[i];
    }
}

int main(int argc, char *argv[])
{
    //function declaration
    void cleanup(void);

    //code
    //allocate host memory
    hostInput1 = (float *)malloc(inputLength * sizeof(float));
    if(hostInput1 == NULL)
    {
        printf("CPU Memory Fatal Error = Can Not Allocate Enough Memory For Host Input Array 1.\nExiting ...\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    hostInput2 = (float *)malloc(inputLength * sizeof(float));
    if(hostInput2 == NULL)
    {
        printf("CPU Memory Fatal Error = Can Not Allocate Enough Memory For Host Input Array 2.\nExiting ...\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    hostOutput = (float *)malloc(inputLength * sizeof(float));
    if(hostOutput == NULL)
    {
        printf("CPU Memory Fatal Error = Can Not Allocate Enough Memory For Host Output Array.\nExiting ...\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    //fill above input host vectors with arbitary but hard-coded data
    hostInput1[0] = 101.0f;
    hostInput1[1] = 102.0f;
    hostInput1[2] = 103.0f;
    hostInput1[3] = 104.0f;
    hostInput1[4] = 105.0f;
    
    hostInput2[0] = 201.0f;
    hostInput2[1] = 202.0f;
    hostInput2[2] = 203.0f;
    hostInput2[3] = 204.0f;
    hostInput2[4] = 205.0f;

    //allocate the device memory
    int size = inputLength * sizeof(float);
    hipError_t err = hipSuccess;
    err = hipMalloc((void **)&deviceInput1, size);
    if(err != hipSuccess)
    {
        printf("GPU Memory Fatal Error = %s In File Name %s At Line No %d.\nExiting ...\n", hipGetErrorString(err), __FILE__, __LINE__);
        cleanup();
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **)&deviceInput2, size);
    if(err != hipSuccess)
    {
        printf("GPU Memory Fatal Error = %s In File Name %s At Line No %d.\nExiting ...\n", hipGetErrorString(err), __FILE__, __LINE__);
        cleanup();
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **)&deviceOutput, size);
    if(err != hipSuccess)
    {
        printf("GPU Memory Fatal Error = %s In File Name %s At Line No %d.\nExiting ...\n", hipGetErrorString(err), __FILE__, __LINE__);
        cleanup();
        exit(EXIT_FAILURE);
    }

    //copy host memory contents to device memory 
    err = hipMemcpy(deviceInput1, hostInput1, size, hipMemcpyHostToDevice);
    if(err != hipSuccess)
    {
        printf("GPU Memory Fatal Error = %s In File Name %s At Line No %d.\nExiting ...\n", hipGetErrorString(err), __FILE__, __LINE__);
        cleanup();
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(deviceInput2, hostInput2, size, hipMemcpyHostToDevice);
    if(err != hipSuccess)
    {
        printf("GPU Memory Fatal Error = %s In File Name %s At Line No %d.\nExiting ...\n", hipGetErrorString(err), __FILE__, __LINE__);
        cleanup();
        exit(EXIT_FAILURE);
    }

    //cuda kernel configuration
    dim3 DimGrid = dim3(ceil(inputLength / 256.0), 1, 1);
    dim3 DimBlock = dim3(256, 1, 1);
    vecAdd<<<DimGrid, DimBlock>>>(deviceInput1, deviceInput2, deviceOutput, inputLength);

    //copy device memory to host memory
    err = hipMemcpy(hostOutput, deviceOutput, size, hipMemcpyDeviceToHost);
    if(err != hipSuccess)
    {
        printf("GPU Memory Fatal Error = %s In File Name %s At Line No %d.\nExiting ...\n", hipGetErrorString(err), __FILE__, __LINE__);
        cleanup();
        exit(EXIT_FAILURE);
    }

    //result
    int i;
    for(i = 0; i < inputLength; i++)
    {
        printf("%f + %f = %f\n", hostInput1[i], hostInput2[i], hostOutput[i]);
    }

    //total cleanup
    cleanup();

    return (0);
}

void cleanup(void)
{
    //code

    //free allocated device memory
    if(deviceOutput)
    {
        hipFree(deviceOutput);
        deviceOutput = NULL;
    }

    if(deviceInput2)
    {
        hipFree(deviceInput2);
        deviceInput2 = NULL;
    }

    if(deviceInput1)
    {
        hipFree(deviceInput1);
        deviceInput1 = NULL;
    }

    //free allocated host memory
    if(hostOutput)
    {
        free(hostOutput);
        hostOutput = NULL;
    }

    if(hostInput2)
    {
        free(hostInput2);
        hostInput2 = NULL;
    }

    if(hostInput1)
    {
        free(hostInput1);
        hostInput1 = NULL;
    }
}

